/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/impl/AuxIndexStructures.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/BroadcastSum.cuh>
#include <faiss/gpu/impl/TinyEigh.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/Limits.cuh>
#include <faiss/gpu/utils/MatrixMult.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <algorithm>
#include <memory>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

// #include "cusolver_utils.h"

namespace faiss {
namespace gpu {

template <typename T>
void runTinyEigh(GpuResources* res,hipStream_t stream,
                 Tensor<T, 3, true>& covMat,
                 Tensor<T, 3, true>& eigVecs,
                 Tensor<T, 2, true>& eigVals){
  
  // thrust::fill(thrust::cuda::par.on(stream),
  //              eigVecs.data(),eigVecs.end(),
  //              Limits<float>::getMax());
  // thrust::fill(thrust::cuda::par.on(stream),
  //              eigVals.data(),eigVals.end(),
  //              Limits<float>::getMax());
  int num = covMat.getSize(0);
  int dim = covMat.getSize(1);

  //
  // -- init --
  //

  hipsolverSyevjInfo_t syevj_params = NULL;
  hipsolverHandle_t cusolverH;
  hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

  float *d_work = nullptr;
  int lwork = 0;
  // int info_gpu = 0;
  std::vector<int> info_gpu(num, 0); 
  int* d_info = nullptr;

  // -- solver handle --
  CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));
  CUDA_VERIFY(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

  // -- error handling  --
  CUDA_VERIFY(hipMalloc(reinterpret_cast<void **>(&d_info), num*sizeof(int)));


  // -- precision --
  float residual = 0;
  int executed_sweeps = 0;
  const float tol = 1.e-10;
  const int max_sweeps = 100;
  const int sort_eig = 1;
  CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));
  CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params, tol));
  // CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps));
  CUSOLVER_CHECK(hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig));



  // -- compute spectrum --
  float* d_A = (float*)covMat.data();
  float* d_W = (float*)eigVals.data();
  int lda = dim;
  
  // CUSOLVER_CHECK(hipsolverDnDsyevjBatched_bufferSize(cusolverH, jobz, uplo, dim,\
  //                                                   d_A, lda, d_W, &lwork, \
  //                                                   syevj_params,num));
  // CUDA_VERIFY(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(float) * lwork));
  // CUSOLVER_CHECK(hipsolverDnDsyevjBatched(cusolverH, jobz, uplo, dim, d_A,\
  //                                        lda, d_W, d_work, lwork, d_info,\
  //                                        syevj_params,num));

  CUSOLVER_CHECK(hipsolverDnSsyevjBatched_bufferSize(cusolverH, jobz, uplo, dim,\
                                                    d_A, lda, d_W, &lwork, \
                                                    syevj_params,num));
  CUDA_VERIFY(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(float) * lwork));
  CUSOLVER_CHECK(hipsolverDnSsyevjBatched(cusolverH, jobz, uplo, dim, d_A,\
                                         lda, d_W, d_work, lwork, d_info,\
                                         syevj_params,num));

  // /* step 4: query working space of syevj */
  // CUSOLVER_CHECK(hipsolverDnDsyevj_bufferSize(cusolverH, jobz, uplo, m,
  //                                            d_A, lda, d_W, &lwork, syevj_params));
  /* step 5: compute eigen-pair   */
  // CUSOLVER_CHECK(hipsolverDnDsyevj(cusolverH, jobz, uplo, m, d_A, lda, d_W,
  //                                 d_work, lwork, devInfo,syevj_params));


  CUDA_VERIFY(hipStreamSynchronize(stream));


  // -- eror handling --
  // CUDA_VERIFY(hipMemcpyAsync(info_gpu.data(), d_info, num*sizeof(int), \
  //                             hipMemcpyDeviceToHost, stream));
  // CUDA_VERIFY(hipStreamSynchronize(stream));
  // for (int index = 0; index < 10; ++index){
  //   if (0 == info_gpu[index]) {
  //     printf("syevj converges \n");
  //   } else if (0 > info_gpu[index]) {
  //     printf("%d-th parameter is wrong \n", -info_gpu[index]);
  //     exit(1);
  //   } else {
  //     printf("WARNING: info = %d : syevj does not converge \n", info_gpu[index]);
  //   }
  // }

  //
  // -- clean up --
  //

  CUSOLVER_CHECK(hipsolverDnDestroySyevjInfo(syevj_params));
  CUSOLVER_CHECK(hipsolverDnDestroy(cusolverH));

}


void runTinyEigh(GpuResources* res,
                 hipStream_t stream,
                 Tensor<double, 3, true>& covMat,
                 Tensor<double, 3, true>& eigVecs,
                 Tensor<double, 2, true>& eigVals){
  runTinyEigh<double>(res,stream,covMat,eigVecs,eigVals);
}

void runTinyEigh(GpuResources* res,
                 hipStream_t stream,
                 Tensor<float, 3, true>& covMat,
                 Tensor<float, 3, true>& eigVecs,
                 Tensor<float, 2, true>& eigVals){
  runTinyEigh<float>(res,stream,covMat,eigVecs,eigVals);
}

void runTinyEigh(GpuResources* res,
                 hipStream_t stream,
                 Tensor<half, 3, true>& covMat,
                 Tensor<half, 3, true>& eigVecs,
                 Tensor<half, 2, true>& eigVals){
  runTinyEigh<half>(res,stream,covMat,eigVecs,eigVals);
}
  

}
}