#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/StaticUtils.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/BurstPatchSearch.cuh>
#include <faiss/gpu/utils/ConversionOperators.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/Float16.cuh>
#include <faiss/gpu/utils/MathOperators.cuh>
#include <faiss/gpu/utils/PtxUtils.cuh>
#include <faiss/gpu/utils/Reductions.cuh>

#include <algorithm>

namespace faiss {
namespace gpu {

#define inline_min(a, b) ( (a) < (b) ? (a) : (b) )
#define inline_max(a, b) ( (a) > (b) ? (a) : (b) )
// #define macro_max(a, b) (a > b)? a: b 
#define legal_access(a, b, c, d) ((((a) >= (c)) && (((a)+(b)) < (d))) ? true : false)

template <typename T, int BatchQueries>
__global__ void burstPatchFillKernel(
	Tensor<T, 4, true, int> burst,
	Tensor<T, 6, true, int> patches,
	Tensor<int, 2, true, int> inds,
    int queryStart, int queryStride,
    int ws, int wb, int wf,
    int dimPerThread, int queriesPerBlock){

    // get the cuda vars 
    int numWarps = utils::divUp(blockDim.x, kWarpSize); 
    int laneId = getLaneId();
    int threadId = threadIdx.x;

    // Unpack Shapes
    int numQueries = patches.getSize(0);
    int k = patches.getSize(1);
    int pt = patches.getSize(2);
    int c = patches.getSize(3);
    int ps = patches.getSize(4);
    int ws2 = ws * ws;
    int nWt = wb + wf + 1;

    // Unpack Shapes [Burst]
    int nframes = burst.getSize(0);
    int height = burst.getSize(2);
    int width = burst.getSize(3);
    int wsHalf = (ws-1)/2;
    int npix = height * width;

    for (int d_index = 0; d_index < dimPerThread; ++d_index){
      for (int qpb = 0; qpb < queriesPerBlock; ++qpb){
#pragma unroll
        for (int qidx = 0; qidx < BatchQueries; ++qidx){

          // compute start of copies 
          int queryIndexStart = queriesPerBlock*BatchQueries*(blockIdx.x);
          // [Query Index]
          int queryIndex = (queryIndexStart + qidx + qpb);
          int queryPix = queryStride*(queryIndex + queryStart);
          if (queryIndex < numQueries){
    
              //
              // [Patch] Indices
              //
    
              int pindex = dimPerThread * threadIdx.x + d_index;
              int denom = 1;
              int kIndex = (pindex) % k;
              denom = k;
              int ptIndex = (pindex / denom) % pt;
              denom  = k * pt;
              int cIndex = (pindex / denom) % c;
              denom  = k * pt * c;
              int hIndex = (pindex / denom) % ps;
              denom  = k * pt * c * ps;
              int wIndex = (pindex / denom) % ps;
              // printf("pi,k,pt,c,h,w: %d,%d,%d,%d,%d,%d\n",
              //        pindex,kIndex,ptIndex,cIndex,hIndex,wIndex);

              // [Ref] Location
              int r_frame = queryPix / npix;
              int r_query_row = (queryPix % npix) / width;
              int r_query_col = (queryPix % npix) % width;
              int r_rowTop = r_query_row - ps/2;
              int r_colLeft = r_query_col - ps/2;
    
              // Frame Offsets
              int shift_t_min = inline_min(0,r_frame - wb);
              int shift_t_max = inline_max(0,r_frame + wf - nframes + pt);
              int shift_t = shift_t_min + shift_t_max;
              int frame_min = inline_max(r_frame - wb - shift_t,0);
              int frame_min_shift = r_frame - frame_min;
    
              // Search Space Offsets
              int spaceIndex = inds[queryIndex][kIndex];
              int frame_index = spaceIndex % nWt;
              int space_row = ((spaceIndex / nWt) / ws) - wsHalf;
              int space_col = ((spaceIndex / nWt) % ws) - wsHalf;
    
              // [Proposed] Location [top-left of search patch]
              int p_frame = r_frame + frame_index - frame_min_shift;
              int p_rowTop = r_rowTop + space_row;
              int p_colLeft = r_colLeft + space_col;
    
              //
              // [Burst] Indices
              //
    
              int tIndex = p_frame;
              int b_hIndex = p_rowTop + hIndex;
              int b_wIndex = p_colLeft + wIndex;
              b_hIndex = (b_hIndex < height) ?  b_hIndex : (2*height - b_hIndex - 1);
              b_wIndex = (b_wIndex < width) ? b_wIndex : (2*width - b_wIndex - 1);
              b_hIndex = (b_hIndex >= 0) ? b_hIndex : (-b_hIndex-1);
              b_wIndex = (b_wIndex >= 0) ? b_wIndex : (-b_wIndex-1);
    
              // [Fill] Patches with "val"
    
              // T val = burst[0][0][0][0];
              // T val = burst[tIndex][cIndex][0][0];
              // T val = burst[0][cIndex][b_hIndex][b_wIndex];
              T val = burst[tIndex][cIndex][b_hIndex][b_wIndex];
              // T val = threadId * 1.0;
              // T val = (T)pindex;
              // patches[queryIndex][kIndex][ptIndex][cIndex][hIndex][wIndex] = val;
              patches[queryIndex][kIndex][ptIndex][cIndex][hIndex][wIndex] = val;
          }
        }
      }
    }
}

template <typename T>
void fillBurstPatches(Tensor<T, 4, true>& burst,
                      Tensor<T, 6, true>& patches,
                      Tensor<int, 2, true>& inds,
                      int queryStart, int queryStride,
                      int ws, int wb, int wf,
                      hipStream_t stream){

  // batching 
  constexpr int batchQueries = 4;

  // unpack shapes 
  int maxThreads = (int)getMaxThreadsCurrentDevice();
  int numQueries = inds.getSize(0); // == numPatches
  int k = inds.getSize(1);
  int ps = patches.getSize(5);
  int c = burst.getSize(1);

  // compute num threads
  int dimPerThread = ps; // how much does each thread handle
  int threadsPerPatch = c*ps; // assuming patchsize_dim = ps*ps; ps*ps / dimPerThread
  int patchesPerQuery = k;
  int queriesPerBlock = 1;// a function of "k"; smaller "k" -> greater "qpb"
  int numThreads = threadsPerPatch * patchesPerQuery * queriesPerBlock;

  // unpack shape of queries
  int nq = patches.getSize(0);
  int pk = patches.getSize(1);
  int pt = patches.getSize(2);
  int pc = patches.getSize(3);
  int ps1 = patches.getSize(4);
  int ps2 = patches.getSize(5);
  fprintf(stdout,"pshape = (%d,%d,%d,%d,%d,%d)\n",nq,pk,pt,pc,ps1,ps2);

  // get grids and threads 
  int numQueryBlocks = (numQueries-1) / (batchQueries*queriesPerBlock) + 1;
  auto grid = dim3(numQueryBlocks);
  auto block = dim3(numThreads);
  fprintf(stdout,"numQueryBlocks,numThreads: %d,%d\n",numQueryBlocks,numThreads);
  fprintf(stdout,"k,ps: %d,%d\n",k,ps);

  burstPatchFillKernel<T,batchQueries>
    <<<grid, block, 0, stream>>>(burst, patches, inds,
                                 queryStart, queryStride,
                                 ws, wb, wf, dimPerThread, queriesPerBlock);
    
  CUDA_TEST_ERROR();
}

void fillBurstPatches(Tensor<float, 4, true>& burst,
                      Tensor<float, 6, true>& patches,
                      Tensor<int, 2, true>& inds,
                      int queryStart, int queryStride,
                      int ws, int wb, int wf,
                      hipStream_t stream){
  fillBurstPatches<float>(burst,patches,inds,queryStart,queryStride,ws,wb,wf,stream);
}

void fillBurstPatches(Tensor<half, 4, true>& burst,
                      Tensor<half, 6, true>& patches,
                      Tensor<int, 2, true>& inds,
                      int queryStart, int queryStride,
                      int ws, int wb, int wf,
                      hipStream_t stream){
  fillBurstPatches<half>(burst,patches,inds,queryStart,queryStride,ws,wb,wf,stream);
}

} // namespace gpu
} // namespace faiss
