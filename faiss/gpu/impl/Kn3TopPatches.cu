#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/impl/AuxIndexStructures.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/BroadcastSum.cuh>
#include <faiss/gpu/impl/Kn3TopPatches.cuh>
#include <faiss/gpu/impl/BurstPatchSearch.cuh>
#include <faiss/gpu/impl/DistanceUtils.cuh>
#include <faiss/gpu/impl/L2Norm.cuh>
#include <faiss/gpu/impl/L2Select.cuh>
#include <faiss/gpu/utils/BlockSelectKernel.cuh>
#include <faiss/gpu/utils/BurstNnfSimpleBlockSelect.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/Limits.cuh>
#include <faiss/gpu/utils/MatrixMult.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <algorithm>
#include <memory>

namespace faiss {
namespace gpu {

template <typename T>
void runKn3TopPatches(GpuResources* res,hipStream_t stream,
                    int ps, int pt, int wf, int wb, int ws,
                    int queryStart, int queryStride,
                    Tensor<T, 4, true>& srch_burst,
                    Tensor<T, 4, true>& fflow,
                    Tensor<T, 4, true>& bflow,
                    Tensor<float, 2, true>& outDistances,
                    Tensor<int, 2, true>& outIndices) {

    // The size of the image burst
    auto nchnls = srch_burst.getSize(0);
    auto nframes = srch_burst.getSize(1);
    auto height = srch_burst.getSize(2);
    auto width = srch_burst.getSize(3);

    // The # of queries; we batch over these twice
    // auto numQueries = queries.getSize(0);
    // auto qdim = queries.getSize(1);

    // The "k" of the knn

    auto k = outDistances.getSize(1);
    auto numQueries = outDistances.getSize(0);

    // The dimensions of the vectors to consider
    // FAISS_ASSERT(qdim == 3);
    FAISS_ASSERT(outDistances.getSize(0) == numQueries);
    FAISS_ASSERT(outIndices.getSize(0) == numQueries);

    // If we're querying against a 0 sized set, just return empty results
    // thrust::fill(thrust::cuda::par.on(stream),
    //              outDistances.data(),
    //              outDistances.end(),
    //              Limits<float>::getMax());
    // thrust::fill(thrust::cuda::par.on(stream),
    //              outIndices.data(),
    //              outIndices.end(),-1);

    // By default, aim to use up to 512 MB of memory for the processing, with
    // both number of queries and number of centroids being at least 512.
    int numSearch = ws*ws;
    int timeWindowSize = wf*wb+1;
    int tileQueries,tileSearch;
    // chooseKn3TileSize(numQueries,numSearch,sizeof(T),tileQueries,tileSearch);
    tileQueries = 4096;
    tileSearch = numSearch;
    // theirs was 512 x 40960
    int numQueryTiles = utils::divUp(numQueries, tileQueries);
    int numSearchTiles = utils::divUp(numSearch, tileSearch);
    // fprintf(stdout,"numQueries,numSearch: %d,%d\n",numQueries,numSearch);
    // fprintf(stdout,"tileQueries,tileSearch: %d,%d\n",tileQueries,tileSearch);
    // fprintf(stdout,"numQueryTiles,numSearchTiles: %d,%d\n",numQueryTiles,numSearchTiles);

    //
    // --> Allocate a frame offsets <--
    //
    // DeviceTensor<float, 2, true> f(
    //         res, makeTempAlloc(AllocType::Other, stream),
    //         {tileQueries, tileSearch * timeWindowSize});
    // DeviceTensor<float, 2, true> distanceBuf2(
    //         res, makeTempAlloc(AllocType::Other, stream),
    //         {tileQueries, tileSearch * timeWindowSize});
    // DeviceTensor<float, 2, true>* distanceBufs[2] = {
    //         &distanceBuf1, &distanceBuf2};


    // We can have any number of vectors to query against, even less than k, in
    // which case we'll return -1 for the index
    FAISS_ASSERT(k <= GPU_MAX_SELECTION_K); // select limitation

    // Temporary output memory space we'll use
    DeviceTensor<float, 2, true> distanceBuf1(
            res, makeTempAlloc(AllocType::Other, stream),
            {tileQueries, tileSearch * timeWindowSize});
    DeviceTensor<float, 2, true> distanceBuf2(
            res, makeTempAlloc(AllocType::Other, stream),
            {tileQueries, tileSearch * timeWindowSize});
    DeviceTensor<float, 2, true>* distanceBufs[2] = {
            &distanceBuf1, &distanceBuf2};

    // DeviceTensor<float, 2, true> outDistanceBuf1(
    //         res, makeTempAlloc(AllocType::Other, stream),
    //         {tileQueries, numSearchTiles * k});
    // DeviceTensor<float, 2, true> outDistanceBuf2(
    //         res, makeTempAlloc(AllocType::Other, stream),
    //         {tileQueries, numSearchTiles * k});
    // DeviceTensor<float, 2, true>* outDistanceBufs[2] = {
    //         &outDistanceBuf1, &outDistanceBuf2};

    // DeviceTensor<int, 2, true> outIndexBuf1(
    //         res, makeTempAlloc(AllocType::Other, stream),
    //         {tileQueries, numSearchTiles * k});
    // DeviceTensor<int, 2, true> outIndexBuf2(
    //         res, makeTempAlloc(AllocType::Other, stream),
    //         {tileQueries, numSearchTiles * k});
    // DeviceTensor<int, 2, true>* outIndexBufs[2] = {
    //         &outIndexBuf1, &outIndexBuf2};

    auto streams = res->getAlternateStreamsCurrentDevice();
    streamWait(streams, {stream});

    int curStream = 0;
    bool interrupt = false;

    // Tile over input queries 1
    for (int i = 0; i < numQueries; i += tileQueries) {
        if (interrupt || InterruptCallback::is_interrupted()) {
            interrupt = true;
            break;
        }
        // fprintf(stdout,"i-loop:[%d]\n",i);

        /*

          -- select correct data view --
          
        */
        int curQuerySize = std::min(tileQueries, numQueries - i);
        auto queryStart_i = queryStart + i;
        auto outDistanceView = outDistances.narrow(0, i, curQuerySize);
        auto outIndexView = outIndices.narrow(0, i, curQuerySize);
        // auto outDistanceBufRowView =
        //         outDistanceBufs[curStream]->narrow(0, 0, curQuerySize);
        // auto outIndexBufRowView =
        //         outIndexBufs[curStream]->narrow(0, 0, curQuerySize);

        // Tile over search-space
        for (int j = 0; j < numSearch; j += tileSearch) {
            if (InterruptCallback::is_interrupted()) {
                interrupt = true;
                break;
            }
            // fprintf(stdout,"j-loop:[%d]\n",j);

            /*

              -- select correct data view --

            */

            int curSearchSize = std::min(tileSearch, numSearch - j);
            int curSearchTile = j / tileSearch;

            int fullSearchSize = curSearchSize * timeWindowSize;
            auto distanceBufView = distanceBufs[curStream]
                                           ->narrow(0, 0, curQuerySize)
                                           .narrow(1, 0, fullSearchSize);
            // auto outDistanceBufColView =
            //         outDistanceBufRowView.narrow(1, k * curSearchTile, k);
            // auto outIndexBufColView =
            //         outIndexBufRowView.narrow(1, k * curSearchTile, k);


            /*

              -- exec kernel --

            */
            
            if (curSearchSize == numSearch){ // we search all at once
              
              thrust::fill(thrust::cuda::par.on(stream),
                           distanceBufView.data(),
                           distanceBufView.end(),
                           Limits<float>::getMax());
              // thrust::fill(thrust::cuda::par.on(stream),
              //              outDistanceView.data(),
              //              outDistanceView.end(),
              //              Limits<float>::getMax());

              runBurstNnfL2Norm(srch_burst,fflow,bflow,
                                queryStart_i,queryStride,
                                distanceBufView,outIndexView,
                                j,curSearchSize,ps,pt,ws,wf,wb,stream);
              runBurstNnfSimpleBlockSelect(distanceBufView,
                                           outDistanceView,
                                           outIndexView,stream);

            }else{ // store in temp bufs

            }
        }

        // curStream = (curStream + 1) % 2;
    }

    // Have the desired ordering stream wait on the multi-stream
    streamWait({stream}, streams);

    if (interrupt) {
        FAISS_THROW_MSG("interrupted");
    }
}



//
// Instantiations of the distance templates
//

void runKn3TopPatches(
        GpuResources* res, hipStream_t stream,
        int ps, int pt, int wf, int wb, int ws,
        int queryStart, int queryStride,
        Tensor<float, 4, true>& srch_burst,
        Tensor<float, 4, true>& fflow,
        Tensor<float, 4, true>& bflow,
        Tensor<float, 2, true>& outDistances,
        Tensor<int, 2, true>& outIndices){
    runKn3TopPatches<float>(res,stream,
                         ps,pt,wf,wb,ws,
                         queryStart,queryStride,
                         srch_burst,fflow,bflow,
                         outDistances,outIndices);
}

void runKn3TopPatches(
        GpuResources* res, hipStream_t stream,
        int ps, int pt, int wf, int wb, int ws,
        int queryStart, int queryStride,
        Tensor<half, 4, true>& srch_burst,
        Tensor<half, 4, true>& fflow,
        Tensor<half, 4, true>& bflow,
        Tensor<float, 2, true>& outDistances,
        Tensor<int, 2, true>& outIndices) {
    runKn3TopPatches<half>(
            res,stream,
            ps,pt,wf,wb,ws,
            queryStart,queryStride,
            srch_burst,fflow,bflow,
            outDistances,outIndices);
}

} // namespace gpu
} // namespace faiss
