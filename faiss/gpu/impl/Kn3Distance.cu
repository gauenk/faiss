#include "hip/hip_runtime.h"
/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <faiss/gpu/GpuResources.h>
#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/impl/AuxIndexStructures.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/impl/BroadcastSum.cuh>
#include <faiss/gpu/impl/Kn3Distance.cuh>
#include <faiss/gpu/impl/DistanceUtils.cuh>
#include <faiss/gpu/impl/L2Norm.cuh>
#include <faiss/gpu/impl/L2Select.cuh>
#include <faiss/gpu/utils/BlockSelectKernel.cuh>
#include <faiss/gpu/utils/DeviceDefs.cuh>
#include <faiss/gpu/utils/Limits.cuh>
#include <faiss/gpu/utils/MatrixMult.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <algorithm>
#include <memory>

namespace faiss {
namespace gpu {

template <typename T>
void runKn3Distance(GpuResources* res,
                    hipStream_t stream,
                    Tensor<T, 4, true>& srch_burst,
                    Tensor<int, 2, true>& queries,
                    Tensor<float, 2, true>& outDistances,
                    Tensor<int, 2, true>& outIndices) {
    // The # of centroids in `centroids` based on memory layout
    auto nchnls = srch_burst.getSize(0);
    auto nframes = srch_burst.getSize(1);
    auto height = srch_burst.getSize(2);
    auto width = srch_burst.getSize(3);

    // The # of queries in `queries` based on memory layout
    auto numQueries = queries.getSize(0);
    auto qdim = queries.getSize(1);


    // The dimensions of the vectors to consider
    FAISS_ASSERT(qdim == 3);
    FAISS_ASSERT(outDistances.getSize(0) == numQueries);
    FAISS_ASSERT(outIndices.getSize(0) == numQueries);

    // If we're querying against a 0 sized set, just return empty results
    fprintf(stdout,"filling.\n");
    thrust::fill(thrust::cuda::par.on(stream),
                 outDistances.data(),
                 outDistances.end(),
                 10.);

    thrust::fill(thrust::cuda::par.on(stream),
                 outIndices.data(),
                 outIndices.end(),
                 2);

    return;

    /*
    // L2: If ||c||^2 is not pre-computed, calculate it
    DeviceTensor<float, 1, true> cNorms;
    if (computeL2 && !centroidNorms) {
        cNorms = DeviceTensor<float, 1, true>(
                res, makeTempAlloc(AllocType::Other, stream), {numCentroids});
        runL2Norm(centroids, centroidsRowMajor, cNorms, true, stream);
        centroidNorms = &cNorms;
    }

    //
    // Prepare norm vector ||q||^2; ||c||^2 is already pre-computed
    //
    DeviceTensor<float, 1, true> queryNorms(
            res, makeTempAlloc(AllocType::Other, stream), {(int)numQueries});

    // ||q||^2
    // if (computeL2) {
    //     runL2Norm(queries, queriesRowMajor, queryNorms, true, stream);
    // }

    // By default, aim to use up to 512 MB of memory for the processing, with
    // both number of queries and number of centroids being at least 512.
    int tileRows = 0;
    int tileCols = 0;
    chooseTileSize(
            numQueries,
            numCentroids,
            dim,
            sizeof(T),
            res->getTempMemoryAvailableCurrentDevice(),
            tileRows,
            tileCols);

    int numColTiles = utils::divUp(numCentroids, tileCols);

    // We can have any number of vectors to query against, even less than k, in
    // which case we'll return -1 for the index
    FAISS_ASSERT(k <= GPU_MAX_SELECTION_K); // select limitation

    // Temporary output memory space we'll use
    DeviceTensor<float, 2, true> distanceBuf1(
            res, makeTempAlloc(AllocType::Other, stream), {tileRows, tileCols});
    DeviceTensor<float, 2, true> distanceBuf2(
            res, makeTempAlloc(AllocType::Other, stream), {tileRows, tileCols});
    DeviceTensor<float, 2, true>* distanceBufs[2] = {
            &distanceBuf1, &distanceBuf2};

    DeviceTensor<float, 2, true> outDistanceBuf1(
            res,
            makeTempAlloc(AllocType::Other, stream),
            {tileRows, numColTiles * k});
    DeviceTensor<float, 2, true> outDistanceBuf2(
            res,
            makeTempAlloc(AllocType::Other, stream),
            {tileRows, numColTiles * k});
    DeviceTensor<float, 2, true>* outDistanceBufs[2] = {
            &outDistanceBuf1, &outDistanceBuf2};

    DeviceTensor<int, 2, true> outIndexBuf1(
            res,
            makeTempAlloc(AllocType::Other, stream),
            {tileRows, numColTiles * k});
    DeviceTensor<int, 2, true> outIndexBuf2(
            res,
            makeTempAlloc(AllocType::Other, stream),
            {tileRows, numColTiles * k});
    DeviceTensor<int, 2, true>* outIndexBufs[2] = {
            &outIndexBuf1, &outIndexBuf2};

    auto streams = res->getAlternateStreamsCurrentDevice();
    streamWait(streams, {stream});

    int curStream = 0;
    bool interrupt = false;

    // Tile over the input queries
    for (int i = 0; i < numQueries; i += tileRows) {
        if (interrupt || InterruptCallback::is_interrupted()) {
            interrupt = true;
            break;
        }

        int curQuerySize = std::min(tileRows, numQueries - i);

        auto outDistanceView = outDistances.narrow(0, i, curQuerySize);
        auto outIndexView = outIndices.narrow(0, i, curQuerySize);

        auto queryView =
                queries.narrow(queriesRowMajor ? 0 : 1, i, curQuerySize);
        auto queryNormNiew = queryNorms.narrow(0, i, curQuerySize);

        auto outDistanceBufRowView =
                outDistanceBufs[curStream]->narrow(0, 0, curQuerySize);
        auto outIndexBufRowView =
                outIndexBufs[curStream]->narrow(0, 0, curQuerySize);

        // Tile over the centroids
        for (int j = 0; j < numCentroids; j += tileCols) {
            if (InterruptCallback::is_interrupted()) {
                interrupt = true;
                break;
            }

            int curCentroidSize = std::min(tileCols, numCentroids - j);
            int curColTile = j / tileCols;

            auto centroidsView = sliceCentroids(
                    centroids, centroidsRowMajor, j, curCentroidSize);

            auto distanceBufView = distanceBufs[curStream]
                                           ->narrow(0, 0, curQuerySize)
                                           .narrow(1, 0, curCentroidSize);

            auto outDistanceBufColView =
                    outDistanceBufRowView.narrow(1, k * curColTile, k);
            auto outIndexBufColView =
                    outIndexBufRowView.narrow(1, k * curColTile, k);

            // L2: distance is ||c||^2 - 2qc + ||q||^2, we compute -2qc
            // IP: just compute qc
            // (query id x dim) x (centroid id, dim)' = (query id, centroid id)
            runMatrixMult(
                    distanceBufView,
                    false, // not transposed
                    queryView,
                    !queriesRowMajor, // transposed MM if col major
                    centroidsView,
                    centroidsRowMajor, // transposed MM if row major
                    computeL2 ? -2.0f : 1.0f,
                    0.0f,
                    res->getBlasHandleCurrentDevice(),
                    streams[curStream]);

            if (computeL2) {
                // For L2 distance, we use this fused kernel that performs both
                // adding ||c||^2 to -2qc and k-selection, so we only need two
                // passes (one write by the gemm, one read here) over the huge
                // region of output memory
                //
                // If we aren't tiling along the number of centroids, we can
                // perform the output work directly
                if (tileCols == numCentroids) {
                    // Write into the final output
                    runL2SelectMin(
                            distanceBufView,
                            *centroidNorms,
                            outDistanceView,
                            outIndexView,
                            k,
                            streams[curStream]);

                    if (!ignoreOutDistances) {
                        // expand (query id) to (query id, k) by duplicating
                        // along rows top-k ||c||^2 - 2qc + ||q||^2 in the form
                        // (query id, k)
                        runSumAlongRows(
                                queryNormNiew,
                                outDistanceView,
                                true, // L2 distances should not go below zero
                                      // due to roundoff error
                                streams[curStream]);
                    }
                } else {
                    auto centroidNormsView =
                            centroidNorms->narrow(0, j, curCentroidSize);

                    // Write into our intermediate output
                    runL2SelectMin(
                            distanceBufView,
                            centroidNormsView,
                            outDistanceBufColView,
                            outIndexBufColView,
                            k,
                            streams[curStream]);

                    if (!ignoreOutDistances) {
                        // expand (query id) to (query id, k) by duplicating
                        // along rows top-k ||c||^2 - 2qc + ||q||^2 in the form
                        // (query id, k)
                        runSumAlongRows(
                                queryNormNiew,
                                outDistanceBufColView,
                                true, // L2 distances should not go below zero
                                      // due to roundoff error
                                streams[curStream]);
                    }
                }
            } else {
                // For IP, just k-select the output for this tile
                if (tileCols == numCentroids) {
                    // Write into the final output
                    runBlockSelect(
                            distanceBufView,
                            outDistanceView,
                            outIndexView,
                            true,
                            k,
                            streams[curStream]);
                } else {
                    // Write into the intermediate output
                    runBlockSelect(
                            distanceBufView,
                            outDistanceBufColView,
                            outIndexBufColView,
                            true,
                            k,
                            streams[curStream]);
                }
            }
        }

        // As we're finished with processing a full set of centroids, perform
        // the final k-selection
        if (tileCols != numCentroids) {
            // The indices are tile-relative; for each tile of k, we need to add
            // tileCols to the index
            runIncrementIndex(
                    outIndexBufRowView, k, tileCols, streams[curStream]);

            runBlockSelectPair(
                    outDistanceBufRowView,
                    outIndexBufRowView,
                    outDistanceView,
                    outIndexView,
                    computeL2 ? false : true,
                    k,
                    streams[curStream]);
        }

        curStream = (curStream + 1) % 2;
    }

    // Have the desired ordering stream wait on the multi-stream
    streamWait({stream}, streams);

    if (interrupt) {
        FAISS_THROW_MSG("interrupted");
    }
    */
}

template <typename T>
void runL2Distance(
        GpuResources* res,
        hipStream_t stream,
        Tensor<T, 4, true>& srch_burst,
        Tensor<int, 2, true>& queries,
        Tensor<float, 2, true>& outDistances,
        Tensor<int, 2, true>& outIndices) {
    runKn3Distance<T>(res,
                      stream,
                      srch_burst,
                      queries,
                      outDistances,
                      outIndices);
}

//
// Instantiations of the distance templates
//

void runL2Distance(
        GpuResources* res,
        hipStream_t stream,
        Tensor<float, 4, true>& srch_burst,
        Tensor<int, 2, true>& queries,
        Tensor<float, 2, true>& outDistances,
        Tensor<int, 2, true>& outIndices){
    runL2Distance<float>(res,
                         stream,
                         srch_burst,
                         queries,
                         outDistances,
                         outIndices);
}

void runL2Distance(
        GpuResources* res,
        hipStream_t stream,
        Tensor<half, 4, true>& srch_burst,
        Tensor<int, 2, true>& queries,
        Tensor<float, 2, true>& outDistances,
        Tensor<int, 2, true>& outIndices) {
    runL2Distance<half>(
            res,
            stream,
            srch_burst,
            queries,
            outDistances,
            outIndices);
}

} // namespace gpu
} // namespace faiss
