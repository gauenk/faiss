#include "hip/hip_runtime.h"
/**
 * Copyright (c) Kent Gauen
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#pragma once

#include <faiss/gpu/utils/DeviceUtils.h>
#include <faiss/gpu/utils/StaticUtils.h>
#include <faiss/gpu/utils/MathOperators.cuh>
#include <faiss/impl/FaissAssert.h>
#include <faiss/gpu/utils/BurstNnfSimpleBlockSelect.cuh>

/****
     Select "topK" from "blockTileSize" of inVals
 ****/

#define ABS(N) (((N)<0)?(-(N)):((N)))

namespace faiss {
  namespace gpu {

    template <int BatchQueries>
    __global__ void burstNnfBlockFill(
	    Tensor<float, 2, true> inVals,
        Tensor<float, 2, true> outVals,
        Tensor<int, 2, true> outKeys){

      int queryIndex = threadIdx.x + blockDim.x * blockIdx.x;
      int numQueries = inVals.getSize(0);
      int numSearch = inVals.getSize(1);
      int k = outVals.getSize(1);
      int compStart = 0;//4000;
      bool legal = queryIndex < numQueries;
      // printf("queryIndex: %d\n",queryIndex);
      // printf("numQueries: %d\n",numQueries);

      if ( legal ) {

        int compIndex = compStart;
        for (int comp = 0; comp < k; ++comp){          
          outVals[queryIndex][comp] = (float)inVals[queryIndex][compIndex];
          outKeys[queryIndex][comp] = (int)compIndex;
          compIndex += 1;
        }
      }
    }

    template <int BatchQueries>
    __global__ void burstNnfBlockSelect(
	    Tensor<float, 2, true> inVals,
        Tensor<float, 2, true> outVals,
        Tensor<int, 2, true> outKeys){

      int queryIndexStart = BatchQueries*(threadIdx.x + blockDim.x * blockIdx.x);
      int numQueries = inVals.getSize(0);
      int numSearch = inVals.getSize(1);
      int k = outVals.getSize(1);
      int kidx = 0;

      for (int qidx = 0; qidx < BatchQueries; ++qidx){
        int queryIndex = queryIndexStart + qidx;
        bool legal = queryIndex < numQueries;
        // printf("queryIndex: %d\n",queryIndex);
        // printf("numQueries: %d\n",numQueries);

        if ( legal ) {

          float outVal_max = outVals[queryIndex][k-1];
          float outVal_curr = outVal_max;

          for (int comp = 0; comp < numSearch; ++comp){

            float inVal = inVals[queryIndex][comp];

            if (inVal < outVal_max){
              kidx = k-1;
              outVal_curr = outVal_max;
              while( inVal < outVal_curr && kidx > 0){
                kidx -= 1;
                outVal_curr = outVals[queryIndex][kidx];
              }
              if (kidx != 0){ kidx += 1; }
              else if (inVal > outVal_curr){ kidx += 1; }

              // shift values up
              for (int sidx = k-1; sidx > kidx; --sidx){
                outVals[queryIndex][sidx] = (float)outVals[queryIndex][sidx-1];
                outKeys[queryIndex][sidx] = (int)outKeys[queryIndex][sidx-1];
              }

              // assign new values
              outVals[queryIndex][kidx] = inVal;
              outKeys[queryIndex][kidx] = (int)comp;
              outVal_max = outVals[queryIndex][k-1];

            }          
          } // for loop over "comp"


          // check for any not set
          // int anyNotSet = k;
          // for (int kidx = 0; kidx < k; ++kidx){
          //   if(outKeys[queryIndex][kidx] < 0){
          //     anyNotSet = (anyNotSet == k) ? kidx : anyNotSet;
          //   }
          //   if(outKeys[queryIndex][kidx] >= numSearch){
          //     anyNotSet = (anyNotSet == k) ? kidx : anyNotSet;
          //   }
          // }

          // fill back with last match; anyNotSet > 0
          // for (int kidx = anyNotSet; kidx < k; ++kidx){
          //   outVals[queryIndex][kidx] = (float)outVals[queryIndex][anyNotSet-1];
          //   outKeys[queryIndex][kidx] = (int)outKeys[queryIndex][anyNotSet-1];
          // }

          // for (int kidx = anyNotSet; kidx < k; ++kidx){
          //   outKeys[queryIndex][kidx] = (int)outKeys[queryIndex][0];
          // }
          // for (int kidx = 0; kidx < k; ++kidx){
          //   outKeys[queryIndex][kidx] = 0;
          // }



        } // if (legal)
      }
    }
    
    void runBurstNnfSimpleBlockSelect(
	Tensor<float, 2, true>& inVals,
	Tensor<float, 2, true>& outVals,
	Tensor<int, 2, true>& outKeys,
	hipStream_t stream){

      // assert shapes 

      // batching
      constexpr int batchQueries = 8;

      // unpack sizes
      int maxThreads = (int) getMaxThreadsCurrentDevice();
      int numQueries = inVals.getSize(0);
      int numSearch = inVals.getSize(1);
      int k = outVals.getSize(1);
      // fprintf(stdout,"numSearch: %d\n",numSearch);
      
      // create num of block x threads
      int numQueryExecs = (numQueries-1) / batchQueries + 1;
      int numQueriesSqrt = (int)(utils::pow(numQueryExecs*1.0, .5)+1);

      // setup kernel params
      auto grid = dim3(numQueriesSqrt);
      auto block = dim3(numQueriesSqrt);

      // launch kernel
      burstNnfBlockSelect<batchQueries>\
        <<<grid, block, 0, stream>>>(inVals, outVals, outKeys);
      // burstNnfBlockFill<<<grid, block, 0, stream>>>(inVals, outVals, outKeys);

      CUDA_TEST_ERROR();

    }
    
  }
}